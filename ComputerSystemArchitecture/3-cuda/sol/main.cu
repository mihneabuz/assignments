#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <vector>

#include "helper.h"

__device__ inline float geoDistanceDevice(float phi1, float theta1, float phi2, float theta2)
{
    float cs = __sinf(phi1) * __sinf(phi2) * __cosf(theta1 - theta2) + __cosf(phi1) * __cosf(phi2);
    cs = max(min(cs, 1.f), -1.f);
    return 6371.f * acos(cs);
}

__global__ void kernel(city* cities, float kmRange, int offsetx, int offsety, int size) {
    const auto x = offsetx + threadIdx.x;
    const auto y = offsety + blockIdx.x;

    if (x >= y || x >= size || y >= size)
        return;

    const auto dist = geoDistanceDevice(cities[x].phi, cities[x].theta,
                                        cities[y].phi, cities[y].theta);

    if (dist <= kmRange) {
        atomicAdd(&(cities[x].reachable), cities[y].pop);
        atomicAdd(&(cities[y].reachable), cities[x].pop);
    }
}

int main(int argc, char** argv) {
    DIE(argc == 1, "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE((argc - 1) % 3 != 0, "./accpop <kmrange1> <file1in> <file1out> ...");

    float kmRange;
    int size, offset[2];
    const int dim = 1024;

    for (int argcID = 1; argcID < argc; argcID += 3) {
        kmRange = atof(argv[argcID]);

        std::vector<city> cities(0);
        readCities(cities, argv[argcID + 1]);
        size = cities.size();
        
        city* citiesDevice;
        hipMalloc(&citiesDevice, cities.size() * sizeof(city));
        hipMemcpy(citiesDevice, &cities.front(), cities.size() * sizeof(city), hipMemcpyHostToDevice);

        offset[0] = 0;
        offset[1] = 0;

        while (offset[0] <= size) {
            while (offset[1] <= size) {
                kernel<<<dim, dim>>>(citiesDevice, kmRange, offset[0], offset[1], size);
                offset[1] += dim;
            }

            offset[0] += dim;
            offset[1] = offset[0];
        }
        hipDeviceSynchronize();
        
        hipMemcpy(&cities[0], citiesDevice, cities.size() * sizeof(city), hipMemcpyDeviceToHost);

        writeCities(cities, argv[argcID + 2]);

        hipFree(citiesDevice);
    }
}
